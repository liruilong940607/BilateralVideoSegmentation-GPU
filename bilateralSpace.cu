#include "hip/hip_runtime.h"
#include "bilateralSpace.h"
#define DATA_SIZE 1048576

int data[DATA_SIZE];
void GenerateNumbers(int *number, int size)
{       
    for(int i = 0; i < size; i++) {
        number[i] = rand() % 10;
    }
}
__global__ static void sumOfSquares(int *num, int* result)
{
    int sum = 0;
    int i;
    for(i = 0; i < DATA_SIZE; i++) {
        sum += num[i] * num[i];
    }
    *result = sum;
}
void run(){
    GenerateNumbers(data, DATA_SIZE);
    int* gpudata, *result;
    hipMalloc((void**) &gpudata, sizeof(int) * DATA_SIZE);
    hipMalloc((void**) &result, sizeof(int));
    hipMemcpy(gpudata, data, sizeof(int) * DATA_SIZE,
            hipMemcpyHostToDevice);
    sumOfSquares<<<1, 1, 0>>>(gpudata, result);
    int sum;
    hipMemcpy(&sum, result, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(gpudata);
    hipFree(result);

    printf("sum: %d/n", sum);
}
